#include "hip/hip_runtime.h"
extern "C" {

#include "pluto.h"
#include "plutoCuda.cuh"

/* ********************************************************************************* */
__device__ void cudaUserDefBoundary (int side, uint3 dataIdx, dim3 dataDim)
/*! 
 *  Assign user-defined boundary conditions.
 *
 * \param [in] side   specifies the boundary side where ghost zones need
 *                    to be filled. It can assume the following 
 *                    pre-definite values: X1_BEG, X1_END,
 *                                         X2_BEG, X2_END, 
 *                                         X3_BEG, X3_END.
 * \param [in] dataIdx
 * \param [in] dataDim
 *
 * The primitive variables of the ghost cells can be set the following way:
 *      cudaV.rho[gpu1D(dataIdx, dataDim)] = ... 
 *      cudaV.vx1[gpu1D(dataIdx, dataDim)] = ... 
 *      cudaV.vx2[gpu1D(dataIdx, dataDim)] = ... (only if COMPONENTS > 1)
 *      cudaV.vx3[gpu1D(dataIdx, dataDim)] = ... (only if COMPONENTS > 2)
 *      cudaV.prs[gpu1D(dataIdx, dataDim)] = ... 
 *
 * The coordinates of the cell can be accessed the following way:
 *      x1 coordinate: cudaGrid.x1[dataIdx.x];
 *      x2 coordinate: cudaGrid.x2[dataIdx.y];
 *      x3 coordinate: cudaGrid.x3[dataIdx.z];
 *
 *********************************************************************************** */
{
    int idx = gpu1D(dataIdx, dataDim);
    real R = cudaGrid.x1[dataIdx.x];
    real phi = cudaGrid.x2[dataIdx.y];
    real sigma_0 = cudaInputParam[SIGMA_REF]
                  *pow(R, -cudaInputParam[ALPHA_SIGMA]);
    
    real xgrid = R * cos(phi);
    real ygrid = R * sin(phi);


    if (side == 0)
    {   
        /* Density floor */
        if (cudaV.rho[idx] < cudaSmallDensity)
        {
            cudaV.rho[idx] = cudaSmallDensity;
        }

        ////////////////////////////////////
        /* Accretion by N-bodies */
        for (int l = 0; l < NB_N; l++)
        {
            real delta_x = xgrid - cudaNb.x[l];
            real delta_y = ygrid - cudaNb.y[l];
            real hill_radius = 0.075;

            // TODO: make this more physical
            real accretion_time_scale = 1/0.069;

            if (delta_x*delta_x + delta_y*delta_y <= hill_radius * hill_radius)
            {
                double acc = cudaV.rho[idx] * cuda_dt / accretion_time_scale;
                // g_dm_planet1 += acc;
                cudaV.rho[idx] -= acc;
            }
        }
        ////////////////////////////////////
        
        
        /* Wave damping */
        real Rmax = cudaDomEnd[IDIR];
        real RoverRmax = R/Rmax;

        /* damping_factor * (orbital period at R_max) */
        real tau = cudaInputParam[DAMPING_FACTOR]
                  *2*CONST_PI
                  *sqrt(Rmax*Rmax*Rmax/(CONST_G_CODE_UNITS*cudaInputParam[M_CO]));

        // if (R >= 0.9*Rmax)
        if (R >= 0.9*Rmax)
        {
            real ramp = 100.0*RoverRmax*RoverRmax - 180*RoverRmax + 81.0;
            real lambda = cuda_dt/tau * ramp;

            cudaV.rho[idx] -= lambda * (cudaV.rho[idx] - sigma_0);
            /* vR_0 = 0.0 */
            cudaV.vx1[idx] -= lambda * (cudaV.vx1[idx]);
        }
    }
    else if (side == X1_BEG)
    {
        int i_act = 2*cudaIBEG - dataIdx.x - 1;
        int offsetIdx = dataIdx.z * dataDim.x * dataDim.y
                       +dataIdx.y * dataDim.x
                       +i_act;

        /* drho / dr = 0 */
        cudaV.vc[RHO][idx] = cudaV.vc[RHO][offsetIdx];

        if (cudaV.vc[VX1][offsetIdx] > 0.0)
            cudaV.vc[VX1][idx] = -cudaV.vc[VX1][offsetIdx];
        else
            cudaV.vc[VX1][idx] = cudaV.vc[VX1][offsetIdx];

        /* domega / dr = 0 */
        cudaV.vc[VX2][idx] =  cudaGrid.x1[dataIdx.x] /cudaGrid.x1[i_act]
                             *cudaV.vc[VX2][offsetIdx];
    }
    else if (side == X1_END)
    {
        cudaV.vc[RHO][idx] = sigma_0;
        cudaV.vc[VX1][idx] = 0.0;
        cudaV.vc[VX2][idx] = sqrt(CONST_G_CODE_UNITS *cudaInputParam[M_CO]/R);
    }
}

#if BODY_FORCE != NO
/* ********************************************************************* */
__device__ real cudaBodyForceVector(int dir, uint3 dataIdx, dim3 dataDim)
/*!
 * Returns the component of the acceleration vector in a given direction.
 * 
 * \param [in] dir      specifies the component of the acceleration vector.
 * \param [in] dataIdx 
 * \param [in] dataDim  
 *
 * The coordinates of the cell can be accessed the following way:
 *      x1 coordinate: cudaGrid.x1[dataIdx.x];
 *      x2 coordinate: cudaGrid.x2[dataIdx.y];
 *      x3 coordinate: cudaGrid.x3[dataIdx.z];
 *
 * The cell-centered primitive variables of the cell can be accessed
 * the following way:
 *      rho: cudaV.rho[gpu1D(dataIdx, dataDim)]
 *      vx1: cudaV.vx1[gpu1D(dataIdx, dataDim)]  
 *      vx2: cudaV.vx2[gpu1D(dataIdx, dataDim)] (only if COMPONENTS > 1)
 *      vx3: cudaV.vx3[gpu1D(dataIdx, dataDim)] (only if COMPONENTS > 2)
 *      prs: cudaV.prs[gpu1D(dataIdx, dataDim)]
 *
 *********************************************************************** */
{
    if (dir == IDIR)
    {
        return 0.0;
    }
    else if (dir == JDIR)
    {
        return 0.0;
    }
    else /* dir == KDIR */
    {
        return 0.0; 

    }
}

/* ********************************************************************* */
__device__ real cudaBodyForcePotential(real x1, real x2, real x3)
/*!
 * Returns the gravitational potential as a function of the coordinates.
 * 
 * \param [in] x1  position in the 1st coordinate direction \f$x_1\f$
 * \param [in] x2  position in the 2nd coordinate direction \f$x_2\f$
 * \param [in] x3  position in the 3rd coordinate direction \f$x_3\f$
 * 
 * \return The body force potential \f$ \Phi(x_1,x_2,x_3) \f$.
 *
 *********************************************************************** */
{
    return 0.0;
}
#endif

#if VISCOSITY != NO
__device__ void cudaViscNu(real *v, double x1, double x2, double x3,
                                    double *nu1, double *nu2)
{
    #if BINARY_H == YES
    real x = x1*cos(x2);
    real y = x1*sin(x2);

    real K = 0.0;
    for (int l = 0; l < CENTRAL_OBJECT; l++)
    {
        real dx = x - cudaNb.x[l]; 
        real dy = y - cudaNb.y[l]; 

        real r3 = dx*dx + dy*dy;
        r3 *= sqrt(r3);

        K += CONST_G_CODE_UNITS*cudaNb.m[l] / r3;
    }
    
    *nu1 =  cudaInputParam[ALPHA_VISC]
           *cudaInputParam[ASPECT_RATIO]*cudaInputParam[ASPECT_RATIO]
           *x1*x1
           *sqrt(K)
           *v[RHO];
    #else 
    *nu1 =  cudaInputParam[ALPHA_VISC]
           *cudaInputParam[ASPECT_RATIO]*cudaInputParam[ASPECT_RATIO]
           *sqrt(CONST_G_CODE_UNITS*cudaInputParam[M_CO]*x1)
           *v[RHO];
    #endif
    *nu2 = 0.0;
}
#endif

#if USE_CUDA_REDUCTION == YES
__device__ void calcReductionValues(real *values, uint3 dataIdx, dim3 dataDim)
{
}
#endif

#if USE_CUDA_ANALYSIS == YES
__device__ void calcAnalysisValues(real *values, uint3 dataIdx, dim3 dataDim)
{
    int idx = gpu1D(dataIdx, dataDim);

    real R = cudaGrid.x1[dataIdx.x];
    real phi = cudaGrid.x2[dataIdx.y];

    real xgrid = R * cos(phi);
    real ygrid = R * sin(phi);

    real sin_phi = sin(cudaGrid.x2[dataIdx.y]);
    real cos_phi = cos(cudaGrid.x2[dataIdx.y]);

    real dV =  cudaGrid.dVx1[dataIdx.x]
              *cudaGrid.dVx2[dataIdx.y]
              *cudaGrid.dVx3[dataIdx.z];

    real sigma = cudaV.rho[idx];
    real v_R = cudaV.vx1[idx];
    real v_phi = cudaV.vx2[idx];

    real dm = sigma * dV;

    real factor_1 = -R*v_R*v_phi
                    /(CONST_G_CODE_UNITS*cudaInputParam[M_CO]);
    real factor_2 =  R*v_phi*v_phi
                    /(CONST_G_CODE_UNITS*cudaInputParam[M_CO]) - 1.0;

    real e_x = -factor_1*sin_phi + factor_2*cos_phi;
    real e_y =  factor_1*cos_phi + factor_2*sin_phi; 

    real e_cell = sqrt(e_x*e_x + e_y*e_y);
    real peri_cell = atan2(e_y, e_x);

    if (peri_cell < 0.0)
        peri_cell += 2*CONST_PI;

    values[AN_MASS] = dm;
    values[AN_E] = dm*e_cell;
    values[AN_PERI] = dm*peri_cell;

    if (R <= cudaInputParam[CUTOFF_RADIUS])
    {
        values[AN_MASS_INNER] = dm;
        values[AN_E_INNER] = dm*e_cell;
        values[AN_PERI_INNER] = dm*peri_cell;
    }
    else 
    {
        values[AN_MASS_INNER] = 0.0;
        values[AN_E_INNER] = 0.0;
        values[AN_PERI_INNER] = 0.0;
    }

    values[AN_SIGMA_MIN] = sigma;

    ////////////////////////////////////
    // output the change in mass
    // TODO: make this more physical
    values[AN_ACC] = 0.0;
    real accretion_time_scale = 1/0.069;
    for (int l = 2; l < NB_N; l++)
        {
            real delta_x = xgrid - cudaNb.x[l];
            real delta_y = ygrid - cudaNb.y[l];
            real hill_radius = 0.075;

            if (delta_x*delta_x + delta_y*delta_y <= hill_radius * hill_radius)
            {
                double acc = cudaV.rho[idx] * cuda_dt / accretion_time_scale;
                // this will be total accretion
                values[AN_ACC] = acc * dV/cuda_dt;
            }
        }
    ////////////////////////////////////
}
#endif

#if USE_CPU_CALC_PARAMETERS == YES
void calcCpuParams(real *values)
{
}
#endif

} /* extern "C" */
