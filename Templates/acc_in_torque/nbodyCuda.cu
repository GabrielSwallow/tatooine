#include "hip/hip_runtime.h"
extern "C" {

#include "pluto.h"
#include "plutoCuda.cuh"

__device__ real cudaNbodySmoothingSquared(int l, uint3 dataIdx, dim3 dataDim)
{
    real R = cudaGrid.x1[dataIdx.x];
    return G_SMOOTHING*G_SMOOTHING
          *cudaInputParam[ASPECT_RATIO]
          *cudaInputParam[ASPECT_RATIO]
          *R*R;
}

} /* extern "C" */
